
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void cuda_hello(void) {
    printf("Hello World from GPU! on Thread ID:%d \n", threadIdx.x);
}

int main(void) {
    cuda_hello<<<1, 10>>>();
    hipDeviceReset();
    return 0;
}

/*The qualifier __global__ tells the compiler that the function will be called from the CPU and exe-
cuted on the GPU. */

/*Triple angle brackets mark a call from the host thread to the code on the device side. A kernel is
executed by an array of threads and all threads run the same code. The parameters within the triple
angle brackets are the execution confi guration, which specifies how many threads will execute the
kernel. In this example, you will run 10 GPU threads.*/
